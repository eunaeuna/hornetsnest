/**
 * @brief
 * @author Oded Green                                                       <br>
 *   Georgia Institute of Technology, Computational Science and Engineering <br>                   <br>
 *   ogreen@gatech.edu
 * @date August, 2017
 * @version v2
 *
 * @copyright Copyright © 2017 Hornet. All rights reserved.
 *
 * @license{<blockquote>
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * * Neither the name of the copyright holder nor the names of its
 *   contributors may be used to endorse or promote products derived from
 *   this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * </blockquote>}
 *
 * @file
 */
#include "Static/PageRank/PageRank.cuh"
#include "PageRankOperators.cuh"

#include <cub.cuh>

namespace hornet_alg {

StaticPageRank::StaticPageRank(HornetGPU& hornet,
                               int  iteration_max,
                               pr_t threshold,
                               pr_t damp) :
                                    StaticAlgorithm(hornet),
                                    load_balacing(hornet) {
    setInputParameters(iteration_max, threshold, damp);
	hd_prdata().nV = hornet.nV();
	gpu::allocate(hd_prdata().prev_pr,  hornet.nV() + 1);
	gpu::allocate(hd_prdata().curr_pr,  hornet.nV() + 1);
	gpu::allocate(hd_prdata().abs_diff, hornet.nV() + 1);
	gpu::allocate(hd_prdata().contri,   hornet.nV() + 1);
	gpu::allocate(hd_prdata().reduction_out, 1);
	reset();
}

StaticPageRank::~StaticPageRank() {
    release();
}

void StaticPageRank::release() {
    gpu::free(hd_prdata().prev_pr);
	gpu::free(hd_prdata().curr_pr);
	gpu::free(hd_prdata().abs_diff);
    gpu::free(hd_prdata().contri);
	gpu::free(hd_prdata().reduction_out);
    host::free(host_page_rank);
}

void StaticPageRank::reset(){
	hd_prdata().iteration = 0;
}

void StaticPageRank::setInputParameters(int  iteration_max,
                                        pr_t threshold,
                                        pr_t damp) {
	hd_prdata().iteration_max   = iteration_max;
	hd_prdata().threshold       = threshold;
	hd_prdata().damp            = damp;
	hd_prdata().normalized_damp = (1.0f - hd_prdata().damp) /
                                  static_cast<float>(hornet.nV());
}

void StaticPageRank::run() {
	forAllnumV(hornet, InitOperator { hd_prdata });
	hd_prdata().iteration = 0;

	pr_t h_out = hd_prdata().threshold + 1;

	while(hd_prdata().iteration < hd_prdata().iteration_max &&
          h_out > hd_prdata().threshold) {

		forAllnumV(hornet, ResetCurr { hd_prdata });
		forAllVertices(hornet, ComputeContribuitionPerVertex { hd_prdata });
		forAllEdges(hornet, AddContribuitionsUndirected { hd_prdata },
                    load_balacing);
		//forAllEdges(hornet, AddContribuitions { hd_prdata }, load_balacing);
		forAllnumV(hornet, DampAndDiffAndCopy { hd_prdata });

		forAllnumV(hornet, Sum { hd_prdata });
		hd_prdata.sync();

        host::copyFromDevice(hd_prdata().reduction_out, h_out);
		hd_prdata().iteration++;
	}
}

void StaticPageRank::printRankings() {
    pr_t*  d_scores, *h_scores;
    vid_t* d_ids, *h_ids;
    gpu::allocate(d_scores,  hornet.nV());
    gpu::allocate(d_ids,     hornet.nV());
    host::allocate(h_scores, hornet.nV());
    host::allocate(h_ids,    hornet.nV());

    gpu::copyToDevice(hd_prdata().curr_pr, hornet.nV(), d_scores);
	forAllnumV(hornet, SetIds { d_ids });

    pr_t*  d_scores_out;
    vid_t* d_ids_out;
    gpu::allocate(d_scores_out,  hornet.nV());
    gpu::allocate(d_ids_out,     hornet.nV());

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
          d_ids, d_ids_out, d_scores, d_scores_out, hornet.nV());

    hipMalloc(&d_temp_storage, temp_storage_bytes);

    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
          d_scores, d_scores_out, d_ids, d_ids_out, hornet.nV());

#if 1
    host::copyFromDevice(d_ids_out, hornet.nV(), h_ids);
    host::copyFromDevice(d_scores_out, hornet.nV(), h_scores);

        for (int i = 1; i < 11; i++)
        std::cout << "Pr[" << h_ids[hornet.nV()-i] << "]:= " <<  h_scores[hornet.nV()-i] << "\n";
#else
    host::copyFromDevice(d_scores, hornet.nV(), h_scores);
    host::copyFromDevice(d_ids,    hornet.nV(), h_ids);

	for (int i = 0; i < 10; i++)
        std::cout << "Pr[" << h_ids[i] << "]:= " <<  h_scores[i] << "\n";
#endif
    std::cout << std::endl;
	forAllnumV(hornet, ResetCurr { hd_prdata });
	forAllnumV(hornet, SumPr     { hd_prdata });

	pr_t h_out;
    host::copyFromDevice(hd_prdata().reduction_out, h_out);
	std::cout << "              " << std::setprecision(9) << h_out << std::endl;

	gpu::free(d_scores);
	gpu::free(d_ids);
	host::free(h_scores);
	host::free(h_ids);
}

const pr_t* StaticPageRank::get_page_rank_score_host() {
    host::allocate(host_page_rank, hornet.nV());
    host::copyFromDevice(hd_prdata().curr_pr, hornet.nV(), host_page_rank);
    return host_page_rank;
}

int StaticPageRank::get_iteration_count() {
	return hd_prdata().iteration;
}

bool StaticPageRank::validate() {
	return true;//?????????
}

}// hornet_alg namespace
